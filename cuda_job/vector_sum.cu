#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>


//thread 1D
__global__ void addKernal(int * c, const int * a, const int * b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



int main()
{
    const int n = 5;

    const int a[n] = {1, 2, 3, 4, 5};
    const int b[n] = {10, 2, 30, 4, 50};
    int c[n]  = {0};

    int *d_a, *d_b, *d_c;

    // Allocate GPU buffer
    hipMalloc((void**)&d_c, n * sizeof(int));
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    
    // Copy input vectors from host memory to GPU buffer.
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n * sizeof(int), hipMemcpyHostToDevice);


    addKernal <<<1, n>>>(d_c, d_a, d_b);
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);

    printf("[1, 2, 3, 4, 5]\n + \n[10, 2, 30, 4, 50] \n = \n [%d, %d, %d, %d, %d]\n",
        c[0], c[1], c[2], c[3], c[4]);

    return 0;
}